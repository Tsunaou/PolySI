#include "hip/hip_runtime.h"
// JNI
#include <jni.h>
#include "gpu_GPUmm.h"

// CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <exception>
#include <utility>
#include <cstdint>

#include <sys/time.h>

// ====== Vars ======
#define THREADS_PER_BLOCK 512
#define REGULATE_BATCH 1000

static const char* get_error(hipblasStatus_t status) {
    return cublasGetStatusName(status);
}

static const char *get_error(hipError_t err) {
    return hipGetErrorString(err);
}

#define CUDA_ASSERT(expr) \
    do { auto err = (expr); if(err) {std::cerr << __FILE__ ":" << __LINE__ << ":" << get_error(err) << '\n'; std::terminate();} } while(0)

static void
denseSgemm(hipblasHandle_t handle, float *gpu_src, float *gpu_dst, int n) {
    float alpha = 1.0, beta = 0.0;
  CUDA_ASSERT(hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n, n,
        &alpha,
        gpu_src, n,
        gpu_src, n,
        &beta,
        gpu_dst, n));
  CUDA_ASSERT(hipDeviceSynchronize());
  std::cerr << "  [GPU] dense gemm\n";
}

static void
denseSgemm(hipblasHandle_t handle, __half *gpu_src, __half *gpu_dst, int n) {
    __half alpha = 1.0, beta = 0.0;
  CUDA_ASSERT(hipblasHgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n, n, n,
        &alpha,
        gpu_src, n,
        gpu_src, n,
        &beta,
        gpu_dst, n));
  CUDA_ASSERT(hipDeviceSynchronize());
  std::cerr << "  [GPU] dense gemm\n";
}

__global__
static void regulateGPU(float *a, int length) {
  int index = (threadIdx.x + blockIdx.x * blockDim.x) * REGULATE_BATCH;
  //printf("block %d, thread %d, index[%d] => [%f]\n", blockIdx.x, threadIdx.x, index, a[index]);
  for (int i=0; i<REGULATE_BATCH; i++) {
    if (index+i < length) {
      a[index + i] = 2 * (a[index + i] != 0);
    }
  }
}

static void regulate(float *gpu_m, int length, float *cpu_m) {
  int num_blocks = ceil((double)length/THREADS_PER_BLOCK/REGULATE_BATCH);
  regulateGPU<<<num_blocks, THREADS_PER_BLOCK>>>(gpu_m, length);
  CUDA_ASSERT(hipGetLastError());
  CUDA_ASSERT(hipDeviceSynchronize());
}

static bool earlyTermination(hipblasHandle_t handle, float *gpu_m_1, float *gpu_m_2, int length) {
  float* result_1 = (float*) malloc (sizeof(float));
  CUDA_ASSERT(hipblasSasum(handle, length, gpu_m_1, 1 /*?*/, result_1));
  float* result_2 = (float*) malloc (sizeof(float));
  CUDA_ASSERT(hipblasSasum(handle, length, gpu_m_2, 1 /*?*/, result_2));
  if (*result_1 == *result_2) {
    printf("EarlyTermination: %.3f == %.3f\n", *result_1, *result_2);
  }
  return *result_1 == *result_2;
}

static int power(float *cpu_m, int n) {
    static float *gpu_m = nullptr, *gpu_m2 = nullptr;
    static hipblasHandle_t handle_c = nullptr;
    static int matrix_n = 0;

    if (!handle_c) {
        CUDA_ASSERT(hipblasCreate(&handle_c));
    }

    if (matrix_n != n) {
        CUDA_ASSERT(hipFree(gpu_m));
        CUDA_ASSERT(hipFree(gpu_m2));
        CUDA_ASSERT(hipMalloc(&gpu_m, n*n*sizeof(float)));
        CUDA_ASSERT(hipMalloc(&gpu_m2, n*n*sizeof(float)));
        matrix_n = n;
    }

  std::cerr << "[INFO] n=" << n << "\n";


  // (1) copy the matrix to GPU
  CUDA_ASSERT(hipMemcpy(gpu_m, cpu_m, n*n*sizeof(float), hipMemcpyHostToDevice));

  int dense_m = 1;

  // (3.3) dense mm then
  float *gpu_src = gpu_m;
  float *gpu_dst = gpu_m2;

  while(dense_m < n) {
    denseSgemm(handle_c, gpu_src, gpu_dst, n);
    dense_m *= 2;
    regulate(gpu_dst, n*n, cpu_m);
    if(earlyTermination(handle_c, gpu_src, gpu_dst, n*n)) {
      std::cerr << "Early termination, dense_m=" << dense_m << ", n=" << n << "\n";
      break;
    }
    std::swap(gpu_src, gpu_dst);
  }

  // (4) copy the result out
  CUDA_ASSERT(hipMemcpy(cpu_m, gpu_m, n*n*sizeof(float), hipMemcpyDeviceToHost));
  std::cerr << "DONE, DM^" << dense_m << "\n";

  return 0;
}

void Java_gpu_GPUmm_power (JNIEnv *env, jclass cls, jfloatArray jarr, jint jn) {
  float *matrix = (float*) env->GetPrimitiveArrayCritical(jarr, 0);
  power(matrix, jn);
  env->ReleasePrimitiveArrayCritical(jarr, matrix, 0);
}

void Java_gpu_GPUmm_booleanPower
  (JNIEnv *env, jclass cls, jobjectArray jmat, jint jn) {
    float *cpu_mat = new float[(size_t) jn * jn];
    jlongArray *jarrs = new jlongArray[jn];
    long **arrs = new long*[jn];

    for (int i = 0; i < jn; i++) {
        jarrs[i] = (jlongArray)env->GetObjectArrayElement(jmat, i);
        arrs[i] = env->GetLongArrayElements(jarrs[i], nullptr);

        for (int j = 0; j < jn; j++) {
            if (arrs[i][j/64] & (1ULL << (j % 64))) {
                cpu_mat[i * jn + j] = 1;
            } else {
                cpu_mat[i * jn + j] = 0;
            }
        }
        cpu_mat[i * jn + i] = 1;
    }

    power(cpu_mat, jn);

    for (int i = 0; i < jn; i++) {
        for (int j = 0; j < jn; j++) {
            if (cpu_mat[i * jn + j] && i != j) {
                arrs[i][j / 64] |= (1ULL << (j % 64));
            }
        }

        env->ReleaseLongArrayElements(jarrs[i], arrs[i], 0);
    }
}
